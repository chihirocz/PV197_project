
#include <hip/hip_runtime.h>
// write your code into this file

#define TILE_X 3
#define TILE_Y 3
#define TILE_Z 16
#define PADDING 0

__global__ void compute_cell(int* in_array, int* out_array, int dim);

void solveGPU(int **dCells, int dim, int iters)
{
	dim3 threadsPerBlock(TILE_X, TILE_Y, TILE_Z);
	dim3 numBlocks((int)ceil(dim/(float)(TILE_X-2)), (int)ceil(dim/(float)(TILE_Y-2)), (int)ceil(dim/(float)(TILE_Z-2)));
	
	int* result_array;
	hipMalloc((void**)&result_array, dim*dim*dim*sizeof(int));
    int* tmp;
	int* array_in;
	int* array_out;

    array_in = *dCells;
    array_out = (int*)result_array;

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(compute_cell), hipFuncCachePreferShared);

	
	for (int i = 0; i < iters; i++)
	{
		compute_cell<<<numBlocks, threadsPerBlock>>>(array_in, array_out, dim);
		result_array = array_out;
        tmp = array_in;
        array_in = array_out;
        array_out = tmp;
	}

    *dCells = result_array; // result array from loop above
    hipFree(array_out);
}


__global__ void compute_cell(int* in_array, int* out_array, int dim)
{
	// using subsegment of 3x3x32 (+2 bordering) cells from entire cube to use coalesced global mem access, i.e. 9 segments
	__shared__ int tile[TILE_X][TILE_Y][TILE_Z];
	
	//int tx = threadIdx.x;

    int idx_x = blockIdx.x*(blockDim.x-2) + threadIdx.x;
    int idx_y = blockIdx.y*(blockDim.y-2) + threadIdx.y;
    int idx_z = blockIdx.z*(blockDim.z-2) + threadIdx.z;
	
    int border_idx_x = blockIdx.x*blockDim.x + threadIdx.x;
    int border_idx_y = blockIdx.y*blockDim.y + threadIdx.y;
    int border_idx_z = blockIdx.z*blockDim.z + threadIdx.z;
	
	int dim2 = dim*dim;

	// loading non-boundary cells into shared
//	int is_border = 0;
//	is_border += 0;

	// loading tile to shared
	tile[threadIdx.x][threadIdx.y][threadIdx.z] = 0;
	if (idx_x < dim && idx_y < dim && idx_z < dim)
	{
		if (border_idx_x > 0 && border_idx_y > 0 && border_idx_z > 0)
		{
			tile[threadIdx.x][threadIdx.y][threadIdx.z] = in_array[(idx_x*dim2)+(idx_y*dim)+idx_z];
		}
	}

	
	__syncthreads();
	
	
	// neighbourhood computation
	// TODO shuffling functions here
	int result = 0;
	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			for (int k = 0; k < 3; k++)
			{
				result += tile[threadIdx.x+i][threadIdx.y+j][threadIdx.z+k];
			}
		}
	}
	result -= tile[threadIdx.x+1][threadIdx.y+1][threadIdx.z+1];
	__syncthreads();
	
	
	// cell life computation
	if ((result < 4) || (result > 5))
	{
		result = 0;
	}
	else if (result == 5)
	{
		result = 1;
	}
	else
	{
		result = tile[threadIdx.x+1][threadIdx.y+1][threadIdx.z+1];
	}
	__syncthreads();
	
	// TODO redesign to avoid 32-way bank conflict
	out_array[(idx_x*dim2)+(idx_y*dim)+idx_z] = result;
}
