#include "hip/hip_runtime.h"
// write your code into this file

#define TILE_WIDTH 3
#define TILE_LENGTH 32


void solveGPU(int **dCells, int dim, int iters)
{
	dim3 threadsPerBlock(TILE_LENGTH, 1, 1);
	dim3 numBlocks(dim/threadsPerBlock.x, dim/threadsPerBlock.y, dim/threadsPerBlock.z);
	
	int* result_array = (int*) hipMalloc(dim*dim*dim*sizeof(int));
    int* tmp, array_in, aray_out;

    array_in = *dCells;
    array_out = result_array;
	
	for (int i = 0; i < iters; i++)
	{
		compute_cell<<numBlocks, threadPerBlock)>>(array_in, array_out, dim);
        tmp = array_in;
        array_in = array_out;
        array_out = tmp;
	}

    *dCells = array_in; // result array from loop above
    hipFree(result_array);
}


__global__ void compute_cell(int* in_array, int* out_array, int dim)
{
	// using subsegment of 3x3x32 (+2 bordering) cells from entire cube to use coalesced global mem access, i.e. 9 segments
	__shared__ int cell_tile[TILE_LENGTH+2][TILE_WIDTH][TILE_WIDTH];
	__shared__ int results[TILE_LENGTH];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int bz = blockIdx.z;
	int tx = threadIdx.x;

    int idx_x = blockIdx.x*blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y*blockDim.y + threadIdx.y;
    int idx_z = blockIdx.z*blockDim.z + threadIdx.z;
	
	// It is guaranteed that dimension in one line is multiple of 128. There are no residues.
	ushort blocks_in_line = dim/TILE_LENGTH;
	int dim2 = dim*dim;
	
	//for (int i = -1; i < 1; i++)
	//{
		//if ((idx_y < 0) || (idx_y >=dim))
		//{
			//cell_tile[tx+1][i+1][j+1] = 0;
		//}
		//else
		//{
			//for (int j = -1; j < 1; j++)
			//{
				//if ((idx_z < 0) || (idx_z >= dim))
				//{
					//// work out the other bounds
					//cell_tile[tx+1][i+1][j+1] = 0;
				//}
				//else
				//{
					//cell_tile[tx+1][i+1][j+1] = in_array[(tx+bx*blockDim.x+by*dim+bz*dim2];
				//}
			//}
		//}
	//}

    for (int i = -1; i < 2; i++)
    {
        for (int j = -1; j < 2; j++)
        {
            for (int k = -1; k < 2; k++)
            {
                short min = idx_x+i;
                min = min(min, idx_y+j);
                min = min(min, idx_z+k);

                short max = idx_x+i;
                max = max(max, idx_y+j);
                max = max(max, idx_z+k);

                if ((min < 0) || (max >= dim))
                {
                    cell_tile[tx+1+i][j+1][k+1] = 0;
                }
                else
                {
                    // above all this is quite stupid, because a value is stored multiple times into shared
                    // but loading "caps" of the block is guaranteed
                
                    cell_tile[tx+1+i][j+1][k+1] = in_array[tx+i+bx*blockDim.x+(by+j)*dim+(bz+k)*dim2];
                }
            }
        }
    }
	// work out tile bounds for cell_tile[0] and cell_tile[TILE_LENGTH]
	__syncthreads();
	
	
	// neighbourhood computation
	// TODO shuffling functions here
	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			for (int k = 0; k < 3; k++)
			{
				results[tx] += cell_tile[tx+k][i][j];
			}
		}
	}
	results[tx] -= cell_tile[tx+1][1][1];
	
	
	// cell life computation
	if ((results[tx] < 4) || (results[tx] > 5))
	{
		results[tx] = 0;
	}
	else if (results[tx] == 5)
	{
		results[tx] = 1;
	}
	else
	{
		results[tx] = cell_tile[tx+1][1][1];
	}
	
	
	// hipMemcpy() results to global
	// TODO redesign to avoid 16-way bank conflict
	out_array[tx+bx*blockDim.x+by*dim+bz*dim2] = results[tx];
}
